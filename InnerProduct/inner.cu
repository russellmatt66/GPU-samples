/*
Kernel for benchmarking inner product computation using GeForce GTX 960 
*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <fstream>

// 
__global__ void innerProduct(float *sum, const float *a, const float *b, const int N){
    // Assume 1D execution configuration
    int threadNum = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = blockDim.x * gridDim.x;
    float partial = 0.0;

    for (int it = threadNum; it < N; it += nthreads ){
        partial += a[it] * b[it]; // 2 Memory Read + 1 Write
    }

    atomicAdd(sum, partial);
}

// Fill out a and b with random values according to grid-stride method 
__global__ void initRandom(float *a, float* b, const int N, const unsigned long long seed){
    int threadNum = threadIdx.x + blockDim.x * blockIdx.x; // tid or tnum is a better name for idiom
    int nthreads = blockDim.x * gridDim.x;

    // Implement device random number generation with curand 
    hiprandState_t state;
    if (threadNum < N){
        hiprand_init(seed, threadNum, 0, &state);
    }

    for (int it = threadNum; it < N; it += nthreads){
        float aRandomValue = static_cast<float>(hiprand_uniform(&state));
        a[threadNum] = aRandomValue; // 1 Memory Read + 1 Write
        float bRandomValue = static_cast<float>(hiprand_uniform(&state));
        b[threadNum] = bRandomValue; // 1 Memory Read + 1 Write
    }
}

// Error-checking Macro from NVIDIA DLI:GSAC CUDA C/C++ course
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

/* Pass execution configuration size, and array length in via command-line */
int main(int argc, char* argv[]){
    int lshift = std::stoi(argv[1]); // N = 2^(lfshift + 1)
    // Time code using CUDA events
    hipEvent_t start_rand, stop_rand, start_inner, stop_inner;
    float time_rand, time_inner;

    hipEventCreate(&start_rand);
    hipEventCreate(&stop_rand);
    hipEventCreate(&start_inner);
    hipEventCreate(&stop_inner);

    // Declare variables and allocate arrays
    int N = 2<<lshift; // left-shifting 2 twenty-times gives 2^23
    float *a, *b, *device_sum = 0;

    int size = N * sizeof(float);
    checkCuda(hipMallocManaged(&a, size));
    checkCuda(hipMallocManaged(&b, size));
    checkCuda(hipMallocManaged(&device_sum, sizeof(float)));

    // Initialize vectors with random data
    /* Set execution configuration using command-line args */
    int num_blocks, num_threads_per_block; 
    num_threads_per_block = std::stoi(argv[2]);
    num_blocks = N / num_threads_per_block;


    unsigned long long seed = 1234;

    hipEventRecord(start_rand,0);
    initRandom<<<num_blocks, num_threads_per_block>>>(a, b, N, seed);
    hipEventRecord(stop_rand,0);
    hipEventSynchronize(stop_rand);
    hipEventElapsedTime(&time_rand, start_rand, stop_rand);
    // checkCuda(cudaDeviceSynchronize());

    // Call innerProduct Kernel 
    hipEventRecord(start_inner,0);
    innerProduct<<<num_blocks, num_threads_per_block>>>(device_sum, a, b, N);
    hipEventRecord(stop_inner,0);
    hipEventSynchronize(stop_inner);
    hipEventElapsedTime(&time_inner, start_inner, stop_inner);
    // checkCuda(cudaDeviceSynchronize());
    
    /* Write data out to validate */
    std::ofstream output_file;
    output_file.open("innercu.csv", std::ofstream::trunc);
    output_file << "i,a,b" << std::endl;

    for (int i = 0; i < N; i++){
        output_file << i << "," << a[i] << "," << b[i] << std::endl;
    }

    printf("The inner product calculated by the CUDA kernel is %lf\n", *device_sum);

    // Print kernel execution times
    printf("initRandom kernel took %lf milliseconds\n", time_rand);
    printf("innerProduct kernel took %lf milliseconds\n", time_inner);

    // Destroy CUDA Events
    hipEventDestroy(start_rand);
    hipEventDestroy(stop_rand);
    hipEventDestroy(start_inner);
    hipEventDestroy(stop_inner);

    // Free arrays and device_sum
    checkCuda(hipFree(a));
    checkCuda(hipFree(b));
    checkCuda(hipFree(device_sum));
    output_file.close();
}