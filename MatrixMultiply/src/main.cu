// #include "../include/matrix.cu"
// #include "../include/matmul.cu"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <cstdlib>
#include <cstdint>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>

// row-major order
#define IDX2D(i, j, N) (((i)*(N))+(j))

__global__ void InitializeMatrices(float *C, float *A, float *B, const int N, const unsigned long long seed){
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	// Generate random number b/w [0.0, 1.0]
    hiprandState_t state;
    if (tidx < N && tidy < N){
        hiprand_init(seed, tidx, 0, &state);
    }

	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			A[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			B[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			C[IDX2D(i, j, N)] = 0.0;
		}
	}
	return;
}

__global__ void MatMul(float *C, const float *A, const float *B, const int N){ 
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	float sum;
	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			sum = 0.0;
			for (int k = 0; k < N; k++){
				sum += A[IDX2D(i, k, N)] * B[IDX2D(k, j, N)];
			}
			C[IDX2D(i, j, N)] = sum;
		}
	}
	return;
} 

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// TODO - Add timing
int main(int argc, char* argv[]){
	// Accept arguments 
	// int N = atoi(argv[1]); // length of matrix side - CHECK IF THIS IS CAUSING BUG
	uint64_t N = atoi(argv[1]);
	int SM_multiplier_x = atoi(argv[2]); // used for changing number of blocks
	int SM_multiplier_y = atoi(argv[3]);
	int num_threads_per_block_x = atoi(argv[4]);
	int num_threads_per_block_y = atoi(argv[5]);

	// Allocate Matrices on Device
	float *A, *B, *C; // flattened arrays because that is easiest with CUDA

	// checkCuda(cudaMallocManaged(&A, (uint64_t)pow((uint64_t)N,2)*sizeof(float))); // cast b/c of integer overflow 
	// checkCuda(cudaMallocManaged(&B, (uint64_t)pow((uint64_t)N,2)*sizeof(float)));
	// checkCuda(cudaMallocManaged(&C, (uint64_t)pow((uint64_t)N,2)*sizeof(float)));
	checkCuda(hipMallocManaged(&A, ((uint64_t)N)*((uint64_t)N)*sizeof(float))); // cast b/c of integer overflow 
	checkCuda(hipMallocManaged(&B, ((uint64_t)N)*((uint64_t)N)*sizeof(float))); // pow(N,2) not behaving
	checkCuda(hipMallocManaged(&C, ((uint64_t)N)*((uint64_t)N)*sizeof(float)));

	std::cout << "Size of matrices is: " << pow(N,2) << std::endl;

    // Get device attributes 
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	// Define execution configuration
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y);
	dim3 grid_dimensions(numberOfSMs * SM_multiplier_x, numberOfSMs * SM_multiplier_y);

	// Set up timer
	hipEvent_t start_search, stop_search;
    hipEventCreate(&start_search);
    hipEventCreate(&stop_search);
    float time_search;

	// Initialize Matrices
	InitializeMatrices<<<block_dimensions, grid_dimensions>>>(C, A, B, N, 1234); // Magic number at the end is seed for rng
	checkCuda(hipDeviceSynchronize());

	// Perform Matrix Multiplication
	hipEventRecord(start_search, 0);
	MatMul<<<block_dimensions, grid_dimensions>>>(C, A, B, N);
	hipEventRecord(stop_search, 0);
	hipEventSynchronize(stop_search);
	hipEventElapsedTime(&time_search, start_search, stop_search);

	std::cout << "Elapsed time is: " << time_search << " ms" << std::endl;

	// Free data
	hipFree(A);
	hipFree(B);
	hipFree(C);
	return 0;
}