// #include "../include/matrix.cu"
// #include "../include/matmul.cu"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <cstdlib>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>

// row-major order
#define IDX2D(i, j, N) (((i)*(N))+(j))

__global__ void InitializeMatrices(float *C, float *A, float *B, const int N, const unsigned long long seed){
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	// Generate random number b/w [0.0, 1.0]
    hiprandState_t state;
    if (tidx < N && tidy < N){
        hiprand_init(seed, tidx, 0, &state);
    }

	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			A[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			B[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			C[IDX2D(i, j, N)] = 0.0;
		}
	}
	return;
}

__global__ void MatMul(float *C, const float *A, const float *B, const int N){ 
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	int sum;
	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			sum = 0;
			for (int k = 0; k < N; k++){
				sum += A[IDX2D(i, k, N)] * B[IDX2D(k, j, N)];
			}
			C[IDX2D(i, j, N)] = sum;
		}
	}
	return;
} 

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]){
	// Accept arguments 
	int N = atoi(argv[1]); // length of matrix side
	int SM_multiplier_x = atoi(argv[2]); // used for changing number of blocks
	int SM_multiplier_y = atoi(argv[3]);
	int num_threads_per_block_x = atoi(argv[4]);
	int num_threads_per_block_y = atoi(argv[5]);

	// Allocate Matrices on Device
	float *A, *B, *C; // flattened arrays because that is easiest with CUDA

	checkCuda(hipMallocManaged(&A, pow(N,2)*sizeof(float)));
	checkCuda(hipMallocManaged(&B, pow(N,2)*sizeof(float)));
	checkCuda(hipMallocManaged(&C, pow(N,2)*sizeof(float)));

	std::cout << "Size of matrices is: " << pow(N,2) << std::endl;

    // Get device attributes 
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	// Define execution configuration
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y);
	dim3 grid_dimensions(numberOfSMs * SM_multiplier_x, numberOfSMs * SM_multiplier_y);

	// Initialize Matrices
	InitializeMatrices<<<block_dimensions, grid_dimensions>>>(C, A, B, N, 1234); // Magic number at the end is seed for rng
	
	// Perform Matrix Multiplication
	MatMul<<<block_dimensions, grid_dimensions>>>(C, A, B, N);

	return 0;
}