// #include "../include/matrix.cu"
// #include "../include/matmul.cu"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <cstdlib>
#include <cstdint>	
// #include <filesystem>
// #include <fstream>
#include <iostream>
#include <string>
#include <thread>
#include <chrono>

// row-major order
#define IDX2D(i, j, N) (((i)*(N))+(j))

/* Device code */
__global__ void InitializeMatrices(float *C, float *A, float *B, const uint64_t N, const unsigned long long seed){
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	// Generate random number b/w [0.0, 1.0]
    hiprandState_t state;
    if (tidx < N && tidy < N){
        hiprand_init(seed, tidx, 0, &state);
    }

	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			A[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			B[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			C[IDX2D(i, j, N)] = 0.0;
		}
	}
	return;
}

__global__ void MatMul(float *C, const float *A, const float *B, const uint64_t N){ 
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	float sum;
	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			sum = 0.0;
			for (int k = 0; k < N; k++){
				sum += A[IDX2D(i, k, N)] * B[IDX2D(k, j, N)];
			}
			C[IDX2D(i, j, N)] = sum;
		}
	}
	return;
} 

/* Host code */
void hostMatMul(float* C, const float *A, const float *B, const uint64_t N, const int begin, const int end){
    // row-major storage
    float sum;
    for (int i = begin; i < end; i++){ 
        for (int j = begin; j < end; j++){
            sum = 0.0;
            for (int k = 0; k < N; k++){
                sum += A[IDX2D(i, j, N)] * B[IDX2D(i, j, N)];
            }
            C[IDX2D(i, j, N)] = sum;
        }
    }
    return;
}

// This is for zeroing out h_C b/w parallel and sequential CPU run
void hostSetAllZero(float *C, const uint64_t N, const int begin, const int end){
	// row-major storage
    for (int i = begin; i < end; i++){ 
        for (int j = begin; j < end; j++){
            C[IDX2D(i, j, N)] = 0.0;
        }
    }
    return;
}


// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* 
TODO 
(1) Cleanup comments
*/  
int main(int argc, char* argv[]){
	// Accept arguments 
	uint64_t N = atoll(argv[1]);
	int SM_multiplier_x = atoi(argv[2]); // used for changing number of blocks
	int SM_multiplier_y = atoi(argv[3]);
	int num_threads_per_block_x = atoi(argv[4]);
	int num_threads_per_block_y = atoi(argv[5]);

	// Allocate device matrices
	float *A, *B, *C; // flattened arrays because that is easiest with CUDA
    
	uint64_t requested_matrix_memory = N*N*sizeof(float);

	checkCuda(hipMalloc(&A, requested_matrix_memory));
	checkCuda(hipMalloc(&B, requested_matrix_memory));
	checkCuda(hipMalloc(&C, requested_matrix_memory));

	std::cout << "Size of matrices is = " << pow(N,2) << std::endl;

    // Get device attributes 
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	std::cout << "Number of SMs on device = " << numberOfSMs << std::endl;

	// Define execution configuration
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y, 1);
	dim3 grid_dimensions(numberOfSMs * SM_multiplier_x, numberOfSMs * SM_multiplier_y, 1);

	// Set up timer
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;

	// Initialize Matrices
	InitializeMatrices<<<block_dimensions, grid_dimensions>>>(C, A, B, N, 1234); // Magic number at the end is seed for rng
	checkCuda(hipDeviceSynchronize());

	// Allocate host matrices
	float *h_A, *h_B, *h_C;

	h_A = (float*)malloc(requested_matrix_memory);
	h_B = (float*)malloc(requested_matrix_memory);
	h_C = (float*)malloc(requested_matrix_memory);

	checkCuda(hipMemcpy(h_A, A, requested_matrix_memory, hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(h_B, B, requested_matrix_memory, hipMemcpyDeviceToHost));
	checkCuda(hipMemcpy(h_C, C, requested_matrix_memory, hipMemcpyDeviceToHost));

	// Perform Matrix Multiplication
	// GPU kernel, and CPU function, are validated in `../test/validate_matmul.cu`
	// Device
	hipEventRecord(start, 0);
	MatMul<<<grid_dimensions, block_dimensions>>>(C, A, B, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	std::cout << "Elapsed CUDA kernel time is = " << time << " ms" << std::endl;

	// Host Code
	// Parallel
	auto start_host = std::chrono::high_resolution_clock::now();
	std::thread t1(hostMatMul, h_C, h_A, h_B, N, 0, N/8);
	std::thread t2(hostMatMul, h_C, h_A, h_B, N, N/8, N/4);
	std::thread t3(hostMatMul, h_C, h_A, h_B, N, N/4, 3*N/8);
	std::thread t4(hostMatMul, h_C, h_A, h_B, N, 3*N/8, N/2);
	std::thread t5(hostMatMul, h_C, h_A, h_B, N, N/2, 5*N/8);
	std::thread t6(hostMatMul, h_C, h_A, h_B, N, 5*N/8, 3*N/4);
	std::thread t7(hostMatMul, h_C, h_A, h_B, N, 3*N/4, 7*N/8);
	std::thread t8(hostMatMul, h_C, h_A, h_B, N, 7*N/8, N);

	t1.join(); t2.join(); t3.join(); t4.join(); t5.join(); t6.join(); t7.join(); t8.join();

	auto stop_host = std::chrono::high_resolution_clock::now();
	auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(stop_host - start_host).count();

	// std::cout << "Elapsed multi-threaded C++ time is: " << elapsed_time << " ms" << std::endl;
	printf("Elapsed multi-threaded C++ time is = %ld us\n", elapsed_time);
	std::cout << "Number of CPU cores = " << 8 << std::endl; 

	// Free data
	hipFree(A);
	hipFree(B);
	hipFree(C);
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}