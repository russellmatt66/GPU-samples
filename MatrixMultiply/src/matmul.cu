// #include "../include/matrix.cu"
// #include "../include/matmul.cu"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <cstdlib>
#include <cstdint>	
// #include <filesystem>
// #include <fstream>
#include <iostream>
#include <string>
#include <thread>
#include <chrono>

// row-major order
#define IDX2D(i, j, N) (((i)*(N))+(j))

/* Device code */
__global__ void InitializeMatrices(float *C, float *A, float *B, const uint64_t N, const unsigned long long seed){
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	// Generate random number b/w [0.0, 1.0]
    hiprandState_t state;
    if (tidx < N && tidy < N){
        hiprand_init(seed, tidx, 0, &state);
    }

	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			A[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			B[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			C[IDX2D(i, j, N)] = 0.0;
		}
	}
	return;
}

__global__ void MatMul(float *C, const float *A, const float *B, const uint64_t N){ 
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	float sum;
	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			sum = 0.0;
			for (int k = 0; k < N; k++){
				sum += A[IDX2D(i, k, N)] * B[IDX2D(k, j, N)];
			}
			C[IDX2D(i, j, N)] = sum;
		}
	}
	return;
} 

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* 
TODO 
*/  
int main(int argc, char* argv[]){
	// Accept arguments 
	uint64_t N = atoll(argv[1]);
	int SM_multiplier_x = atoi(argv[2]); // used for changing number of blocks
	int SM_multiplier_y = atoi(argv[3]);
	int num_threads_per_block_x = atoi(argv[4]);
	int num_threads_per_block_y = atoi(argv[5]);

	// Allocate device matrices
	float *A, *B, *C; // flattened arrays because that is easiest with CUDA
    
	uint64_t requested_matrix_memory = N*N*sizeof(float);

	checkCuda(hipMalloc(&A, requested_matrix_memory));
	checkCuda(hipMalloc(&B, requested_matrix_memory));
	checkCuda(hipMalloc(&C, requested_matrix_memory));

	std::cout << "Size of matrices is = " << pow(N,2) << std::endl;

    // Get device attributes 
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	std::cout << "Number of SMs on device = " << numberOfSMs << std::endl;

	// Define execution configuration
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y, 1);
	dim3 grid_dimensions(numberOfSMs * SM_multiplier_x, numberOfSMs * SM_multiplier_y, 1);

	// Set up timer
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time;

	// Initialize Matrices
	InitializeMatrices<<<block_dimensions, grid_dimensions>>>(C, A, B, N, 1234); // Magic number at the end is seed for rng
	checkCuda(hipDeviceSynchronize());

	// Perform Matrix Multiplication
	// GPU kernel is validated in `../test/validate_matmul.cu`
	// Device
	hipEventRecord(start, 0);
	MatMul<<<grid_dimensions, block_dimensions>>>(C, A, B, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	std::cout << "Elapsed CUDA kernel time is = " << time << " ms" << std::endl;

	// Free data
	hipFree(A);
	hipFree(B);
	hipFree(C);

	return 0;
}