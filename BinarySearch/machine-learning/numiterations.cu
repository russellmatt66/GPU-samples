#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <string>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <math.h>

#include "binarytree.h"

/*
Code to produce binary for usage in clean.py, so that accurate values of the effective bandwidth can be determined 
*/

// Binary tree functionality for device 
struct d_BTNode {
    int cell;
    int num_iter;
    int num_node;
    d_BTNode* left;
    d_BTNode* right;
};

__device__ d_BTNode* d_createBTNode(int cell, int num_iter, int node){
    d_BTNode* d_newNode;
    (d_BTNode*)hipMalloc(&d_newNode, sizeof(d_BTNode));
    if (d_newNode != NULL){
        d_newNode->cell = cell;
        d_newNode->num_iter = num_iter;
        d_newNode->num_node = node;
        d_newNode->left = NULL;
        d_newNode->right = NULL;
    }
    return d_newNode; 
}

// Build the binary tree
// Step 1: Build all the nodes
// Need functions to get the cell, and number of iterations
__device__ int getCell(int i, int Nx){
    // Base case
    if (i == Nx-2){
        return i;
    }
    return 0;
}

__device__ int getNumIter(int i, int Nx){
    // This is an easy problem
    return 0;
}

__global__ void buildNodes(d_BTNode** all_nodes, int Nx){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    int cell = 0;
    int num_iter = 0;
    for (int i = tidx; i < Nx - 1; i += nthreads){ // number of nodes = Nx-1 = number of cells 
        cell = getCell(i,Nx);
        num_iter = getNumIter(i,Nx);
        all_nodes[i] = d_createBTNode(cell, num_iter, i);
    }

    return;
}

// Step 2: Connect them together
__global__ void connectNodes(d_BTNode** all_nodes, int Nx){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = blockDim.x * gridDim.x;
    
    // Raster through all_nodes and use num_node to 

    return;
}

// These parts are just for reference
// __global__ void d_buildTree(d_BTNode* root, int Nx, int low, int high, int guess, int level){
//     if (root == NULL || level > (int)log2(Nx)){
//         return;
//     }
//     int left_low = low; 
//     int left_high = guess;
//     int left_guess = (left_low + left_high) / 2;
//     int right_low = guess;
//     int right_high = high;
//     int right_guess = (right_low + right_high) / 2;
//     d_BTNode* leftNode = d_createBTNode(left_guess, level);
//     d_BTNode* rightNode = d_createBTNode(right_guess, level);
//     root->left = leftNode;
//     root->right = rightNode;
//     d_buildLeaves(root->left, Nx, left_low, left_high, left_guess, level + 1);
//     d_buildLeaves(root->right, Nx, right_low, right_high, right_guess, level + 1);
// }

// __device__ void d_buildLeaves(d_BTNode* parent, int Nx, int low, int high, int guess, int level){
//     if (parent == NULL || level > (int)log2(Nx)){
//         return;
//     }
//     int left_low = low; 
//     int left_high = guess;
//     int left_guess = (left_low + left_high) / 2;
//     int right_low = guess;
//     int right_high = high;
//     int right_guess = (right_low + right_high) / 2;
//     d_BTNode* leftNode = d_createBTNode(left_guess, level);
//     d_BTNode* rightNode = d_createBTNode(right_guess, level);
//     parent->left = leftNode;
//     parent->right = rightNode;
//     d_buildLeaves(parent->left, Nx, left_low, left_high, left_guess, level + 1);
//     d_buildLeaves(parent->right, Nx, right_low, right_high, right_guess, level + 1);
// }

// 
void writeBST(BTNode* root, int jump, int *counter, std::ofstream& bst_file){
    if (root == NULL){
        return;
    }
    else if ((*counter) % jump == 0){
        bst_file << "Node " << *counter << " represents looking in cell " << root->val << ", where it would take " 
            << root->depth << " iterations to find a particle there" << std::endl; 
    }
    (*counter)++;
    writeBST(root->left, jump, counter, bst_file);
    (*counter)++;
    writeBST(root->right, jump, counter, bst_file);
}

__device__ void getNumIterations(d_BTNode*, int, const int);

// Function to simulate how many total iterations binary search takes to find a given population of particles 
__global__ void simulateSearch(int* sum, const int* num_iters, const int* p_cells, const int N){
    // num_iters 
    // - integer array whose values correspond to how many iterations it takes to find a particle in that cell
    // - size: Nx - 1
    // - the data for the values comes from a binary tree that represents the various outcomes
    // p_cells 
    // - integer array that corresponds to the grid-cells where the individuals of a population of particles can be found
    // - size: N
    // - p_cells[i] \in [0,Nx-2] (there are Nx-1 grid-cells)
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    int partial = 0;

    for (int i = tidx; i < N; i+=nthreads){
        partial += num_iters[p_cells[i]]; // tidy
    }

    atomicAdd(sum, partial);
}

__global__ void initializeNumIters(d_BTNode* root, int* num_iters, const int Nxm1){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    for (int j = tidx; tidx < Nxm1; tidx += nthreads){
        getNumIterations(root, num_iters, j);
    }
}

__device__ void getNumIterations(d_BTNode* root, int* num_iters, const int j){
    // Depth-first search 
    if (root->cell == j){
        num_iters[j] = root->num_iter;
        return;
    }

    if (root->left != NULL){
        getNumIterations(root->left, num_iters, j);
    }
    if (root->right != NULL){
        getNumIterations(root->right, num_iters, j);
    }
    return;
}


__global__ void initializePCells(int* p_cells, const int N, const int Nx){
    int tidx = threadIdx.x + blockDim.x * blockIdx.x; 
    int nthreads = blockDim.x * gridDim.x;

    unsigned long long seed = 1234;

    // Implement device random number generation with hiprand 
    hiprandState_t state;
    if (tidx < N){
        hiprand_init(seed, tidx, 0, &state);
    }

    // Get random integer between [0,Nx-2] to represent which cell the particle is in
    float aRandomValue = 0;
    for (int i = tidx; i < N; i += nthreads){
        aRandomValue = static_cast<float>(hiprand_uniform(&state)) * (Nx-2); 
        p_cells[i] = (int)aRandomValue;
    }
}

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]){
    int N = std::stoi(argv[1]);
    int Nx = std::stoi(argv[2]);

    // Initialize device specific parameters
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    // Create device data
    int *num_iters, *p_cells, *total_iters = 0;


    checkCuda(hipMallocManaged(&num_iters, (Nx-1)*sizeof(int)));
    checkCuda(hipMallocManaged(&p_cells, N*sizeof(int)));
    checkCuda(hipMallocManaged(&total_iters, sizeof(int)));


    // Create binary tree with Nx nodes, representing binary search outcomes
    int low = 0, high = Nx-1;
    int guess = (low + high) / 2;
    int level = 1;
    BTNode* root = createBTNode(guess, level);
    buildLeaves(root, Nx, low, high, guess, level); // can use this to check against device version

    std::ofstream bst_file; 
    bst_file.open("bst.txt");
    int *counter = 0;
    int jump = Nx / 16;
    writeBST(root, jump, counter, bst_file);
    bst_file.close();

    // Create binary tree on device
    d_BTNode* d_root;
    checkCuda(hipMallocManaged(&d_root, sizeof(d_BTNode)));

    d_root->cell = guess;
    d_root->num_iter = 1;
    d_root->num_node = 0; // 0-indexed
    d_root->left = NULL;
    d_root->right = NULL;

    d_BTNode** d_all_bst_nodes;
    checkCuda(hipMallocManaged(&d_all_bst_nodes, Nx*sizeof(d_BTNode)));
    d_all_bst_nodes[0] = d_root;

    // Define execution configuration
    int num_blocks = numberOfSMs;
    int num_threads_per_block = 32;

    // Create bst on device
    // d_buildTree<<<num_blocks, num_threads_per_block>>>(d_root, Nx, low, high, guess, level);
    // STEP 1: BUILD ALL THE NODES
    buildNodes<<<num_blocks, num_threads_per_block>>>(d_all_bst_nodes, Nx);
    checkCuda(hipDeviceSynchronize());
    // STEP 2: CONNECT THEM TOGETHER
    connectNodes<<<num_blocks, num_threads_per_block>>>(d_all_bst_nodes, Nx);
    checkCuda(hipDeviceSynchronize());

    // Initialize num_iters, and p_cells
    initializeNumIters<<<num_blocks, num_threads_per_block>>>(d_root, num_iters, Nx-1); // There are Nx-1 cells 
    initializePCells<<<num_blocks, num_threads_per_block>>>(p_cells, N, Nx);
    checkCuda(hipDeviceSynchronize());

    // SANITY CHECK - Check the values of binary tree, num_iters, and p_cells
    std::ofstream p_cell_file, num_iter_file;

    p_cell_file.open("p_cells.txt");
    jump = N / 16;
    for (int i = 0; i < N; i += jump){
        p_cell_file << "Particle " << i << " is in cell " << p_cells[i] << std::endl;
    }
    p_cell_file.close();

    num_iter_file.open("num_iters.txt");
    jump = Nx / 16;
    for (int j = 0; j < Nx; j += jump){
        num_iter_file << "It takes " << num_iters[j] << " iterations to find a particle in cell " << j << std::endl;
    }
    num_iter_file.close();


    // Call CUDA kernels to simulate the binary search algorithm, and compute total number of iterations required  
    simulateSearch<<<num_blocks, num_threads_per_block>>>(total_iters, num_iters, p_cells, N);
    checkCuda(hipDeviceSynchronize());

    // DO NOT DELETE, clean.py catches this value!
    printf("%d\n", *total_iters / N);

    // Free unified memory
    hipFree(p_cells);
    hipFree(num_iters);
    hipFree(d_root);
    hipFree(d_all_bst_nodes);
    hipFree(total_iters);
    return 0;
}